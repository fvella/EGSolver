/** \file  dev_solver.cu
 * \brief Wrapper per le diverse implementazioni di solver paralleli 
 *
 */

#ifndef FLAGDEV_SOLVER_CU



//#include <hip/hip_runtime.h>

//#include "common.h"
#include "dev_common.h"
#include "dev_solver.h"
#include "utils.cu"

extern char* str;
extern int num_nodi;
extern int max_pesi;
extern int MG_pesi;
extern int num_archi;
extern int counter_nodi0;
extern int max_outdegree;
extern int *host_allData;
extern int *host_csrPtrInSuccLists;
extern int *host_csrSuccLists;
extern int *host_revSuccLists;
extern int *host_csrPesiArchi;
extern int *host_ResNodeValues1;
extern int *host_ResNodeValues2;
extern int *host_ResNodeValuesAux;
extern int *host_flag;
extern int *hdev_allData;
extern int *hdev_csrPtrInSuccLists;
extern int *hdev_csrSuccLists;
extern int *hdev_revSuccLists;
extern int *hdev_csrPesiArchi;
extern int *hdev_ResNodeValues1;
extern int *hdev_ResNodeValues2;
extern int *hdev_ResNodeValuesAux;
extern int *hdev_flag;
extern uint timeout_expired;
extern int *hdev_nodeFlags1;
extern int *hdev_nodeFlags2;

extern int *host_csrDataArchiAux;
extern int *hdev_csrDataArchiAux;

extern int *csrPtrInSuccLists;
extern int *nodePriority;
extern char *nodeOwner;
extern char *nodeFlags;
extern int *csrPesiArchi;
extern int *csrSuccLists;
extern int *revSuccLists;
extern int *nomeExt_of_nomeInt;
extern int *nomeInt_of_nomeExt;
extern int *mapping;
extern int *revmapping;
extern char **nodeName;

extern int *host_transData;
extern int *host_cscPtrInPredLists;
extern int *host_cscPredLists;
extern int *host_cscPesiArchiPred;

extern int *hdev_transData;
extern int *hdev_cscPtrInPredLists;
extern int *hdev_cscPredLists;
extern int *hdev_cscPesiArchiPred;

extern config configuration;
extern stat statistics;


#include "dev_EG_alg.cu"


void gpu_solver() {
	hipEvent_t cuSolveStart, cuSolveStop;
	float solvetime;
	hipEventCreate(&cuSolveStart);
	hipEventCreate(&cuSolveStop);

	hipEventRecord(cuSolveStart, 0);

	switch (configuration.algoritmo) {
		case ALGOR_EG0:  // wrap sulla versione --eg su gpu
			EG_gpu_solver();
			break;
		case ALGOR_EG:
			EG_gpu_solver();
			break;
		default:
			EG_gpu_solver();
			break;
	}

	hipEventRecord(cuSolveStop, 0);
	hipEventSynchronize(cuSolveStop);
	hipEventElapsedTime(&solvetime, cuSolveStart, cuSolveStop);
	statistics.solvingtime = solvetime;
	
	//recupera risultati da device:
	hipDeviceSynchronize();
	CUDASAFE( hipMemcpy(host_ResNodeValues1, hdev_ResNodeValues1, num_nodi*sizeof(int),hipMemcpyDeviceToHost) , "hipMemcpyDeviceToHost host_ResNodeValues1");
	//hipDeviceSynchronize();

}


int copia_dati_su_device() {
        int num_total_mem = num_nodi+1+num_archi+num_archi+num_nodi+num_nodi +1 +1; //(1 per i flag) 
//	printf("\tCOPIA DATI num_nodi=%d\n",num_nodi);fflush(stdout);
	CUDASAFE( hipMemcpy(hdev_allData, host_allData, num_total_mem*sizeof(int),hipMemcpyHostToDevice) , "hipMemcpyHostToDevice dev_allData");

//	printf("\tCOPIATI\n");fflush(stdout);
	return(0);
}


int alloca_memoria_host() {
        int num_total_mem = num_nodi+1+num_archi+num_archi+num_nodi+num_nodi +1 +1; //(1 per i flag)

	num_total_mem += num_archi; // per csrDataArchiAux
	CUDASAFE( hipHostMalloc((void**)&host_allData, num_total_mem*sizeof(int)) , "hipHostMalloc: host_allData[]");
	host_csrPtrInSuccLists = host_allData;
	host_csrSuccLists = host_allData+num_nodi+1;
	host_csrPesiArchi = host_csrSuccLists+num_archi;
	host_ResNodeValues1 = host_csrPesiArchi+num_archi; 
	if ((configuration.algoritmo == ALGOR_EG0)) {
		host_ResNodeValues2 = host_ResNodeValues1+num_nodi; }
	else { host_ResNodeValues2 = host_ResNodeValues1; }
	host_flag = host_ResNodeValues2+num_nodi; 
	host_csrDataArchiAux = host_flag+1; 

	memset(host_ResNodeValues1, 0, num_nodi*sizeof(int)); //azzera vettore risultati

	if ((configuration.algoritmo == ALGOR_EG) || (configuration.algoritmo == ALGOR_EG0)) {
		host_ResNodeValuesAux = NULL;
		// NON USATO host_ResNodeValuesAux = host_csrDataArchiAux + num_archi;   // spazio addizionale per ResNodeValuesAux[]
	} else {
		host_revSuccLists = NULL;
		host_ResNodeValuesAux = NULL;
	}

	if ((configuration.algoritmo == ALGOR_EG) || (configuration.algoritmo == ALGOR_EG0)) {
		int num_trans_mem = num_nodi+1+(2*num_archi);
		//CUDASAFE( hipMalloc((void **)&host_transData, num_trans_mem*sizeof(int)) , "hipMalloc: &host_transData[]");
		host_transData = (int *)malloc(num_trans_mem*sizeof(int));  checkNullAllocation(host_transData,"allocazione host_transData");
		host_cscPtrInPredLists = host_transData;
		host_cscPredLists = host_transData + num_nodi+1;
		host_cscPesiArchiPred = host_cscPredLists + num_archi;
	}

	return(0);
}





int alloca_memoria_device() {
	int num_total_mem = num_nodi+1+num_archi+num_archi+num_nodi+num_nodi +1 +1; //(1 per i flag)

	CUDASAFE( hipMalloc((void **)&hdev_allData, num_total_mem*sizeof(int)) , "hipMalloc: &hdev_allData[]");
	hdev_csrPtrInSuccLists = hdev_allData;
	hdev_csrSuccLists = hdev_allData + num_nodi+1;
	hdev_csrPesiArchi = hdev_csrSuccLists + num_archi;
	hdev_ResNodeValues1 = hdev_csrPesiArchi + num_archi;
	if ((configuration.algoritmo == ALGOR_EG0)) {
		hdev_ResNodeValues2 = hdev_ResNodeValues1 + num_nodi; }
	else { hdev_ResNodeValues2 = hdev_ResNodeValues1; }
	hdev_flag = hdev_ResNodeValues2 + num_nodi;
	hdev_csrDataArchiAux = hdev_flag+1; 

	CUDASAFE( hipMemcpyToSymbol(HIP_SYMBOL(dev_allData), &hdev_allData, sizeof(int *), 0, hipMemcpyHostToDevice) , "hipMemcpyToSymbol dev_allData");
	CUDASAFE( hipMemcpyToSymbol(HIP_SYMBOL(dev_csrPtrInSuccLists), &hdev_csrPtrInSuccLists, sizeof(int *), 0, hipMemcpyHostToDevice) , "hipMemcpyToSymbol dev_csrPtrInSuccLists");
	CUDASAFE( hipMemcpyToSymbol(HIP_SYMBOL(dev_csrSuccLists), &hdev_csrSuccLists, sizeof(int *), 0, hipMemcpyHostToDevice) , "hipMemcpyToSymbol dev_csrSuccLists");
	CUDASAFE( hipMemcpyToSymbol(HIP_SYMBOL(dev_csrPesiArchi), &hdev_csrPesiArchi, sizeof(int *), 0, hipMemcpyHostToDevice) , "hipMemcpyToSymbol dev_csrPesiArchi");
	CUDASAFE( hipMemcpyToSymbol(HIP_SYMBOL(dev_ResNodeValues1), &hdev_ResNodeValues1, sizeof(int *), 0, hipMemcpyHostToDevice) , "hipMemcpyToSymbol dev_ResNodeValues1");
	CUDASAFE( hipMemcpyToSymbol(HIP_SYMBOL(dev_ResNodeValues2), &hdev_ResNodeValues2, sizeof(int *), 0, hipMemcpyHostToDevice) , "hipMemcpyToSymbol dev_ResNodeValues2");
	CUDASAFE( hipMemcpyToSymbol(HIP_SYMBOL(dev_flag), &hdev_flag, sizeof(int *), 0, hipMemcpyHostToDevice) , "hipMemcpyToSymbol dev_flag");
	CUDASAFE( hipMemcpyToSymbol(HIP_SYMBOL(dev_csrDataArchiAux), &hdev_csrDataArchiAux, sizeof(int *), 0, hipMemcpyHostToDevice) , "hipMemcpyToSymbol dev_csrDataArchiAux");

	CUDASAFE( hipMemset(hdev_ResNodeValues1, 0, 2*num_nodi*sizeof(int)) , "hipMemset: hdev_ResNodeValues1e2[]");

	if ((configuration.algoritmo == ALGOR_EG) || (configuration.algoritmo == ALGOR_EG0)) {
		int num_trans_mem = num_nodi+1+(2*num_archi);   
		num_trans_mem += 2*(num_nodi+1);  //2*(num_nodi+1) per nodeFlags1,2
		CUDASAFE( hipMalloc((void **)&hdev_transData, num_trans_mem*sizeof(int)) , "hipMalloc: &hdev_transData[]");
		hdev_cscPtrInPredLists = hdev_transData;
		hdev_cscPredLists = hdev_transData + num_nodi+1;
		hdev_cscPesiArchiPred = hdev_cscPredLists + num_archi;
		hdev_nodeFlags1 = hdev_cscPesiArchiPred + num_archi;
		hdev_nodeFlags2 = hdev_nodeFlags1 + num_nodi +1;

		CUDASAFE( hipMemcpyToSymbol(HIP_SYMBOL(dev_transData), &hdev_transData, sizeof(int *), 0, hipMemcpyHostToDevice) , "hipMemcpyToSymbol dev_transData");
		CUDASAFE( hipMemcpyToSymbol(HIP_SYMBOL(dev_cscPtrInPredLists), &hdev_cscPtrInPredLists, sizeof(int *), 0, hipMemcpyHostToDevice) , "hipMemcpyToSymbol dev_cscPtrInPredLists");
		CUDASAFE( hipMemcpyToSymbol(HIP_SYMBOL(dev_cscPredLists), &hdev_cscPredLists, sizeof(int *), 0, hipMemcpyHostToDevice) , "hipMemcpyToSymbol dev_cscPredLists");
		CUDASAFE( hipMemcpyToSymbol(HIP_SYMBOL(dev_cscPesiArchiPred), &hdev_cscPesiArchiPred, sizeof(int *), 0, hipMemcpyHostToDevice) , "hipMemcpyToSymbol dev_cscPesiArchiPred");
		CUDASAFE( hipMemcpyToSymbol(HIP_SYMBOL(dev_nodeFlags1), &hdev_nodeFlags1, sizeof(int *), 0, hipMemcpyHostToDevice) , "hipMemcpyToSymbol dev_nodeFlags1");
		CUDASAFE( hipMemcpyToSymbol(HIP_SYMBOL(dev_nodeFlags2), &hdev_nodeFlags2, sizeof(int *), 0, hipMemcpyHostToDevice) , "hipMemcpyToSymbol dev_nodeFlags2");

		CUDASAFE( hipMemset(hdev_nodeFlags1, 0, 2*(1+num_nodi)*sizeof(int)) , "hipMemset: hdev_ResNodeValues1e2[]");
	}

	return(0);
}

int dealloca_memoria_host() {
	int inn; 

	CUDASAFE( hipHostFree(host_allData) , "hipHostFree: host_allData[]");
	free(host_transData);
	free(csrPtrInSuccLists);
	free(nodePriority);
	free(nodeOwner);
	free(nodeFlags);
	free(csrPesiArchi);
	free(csrSuccLists);
	free(nomeExt_of_nomeInt);
	free(nomeInt_of_nomeExt);
	free(mapping);
	free(revmapping);
	for (inn=0; inn<num_nodi; inn++) { free(nodeName[inn]); }
	free(nodeName);

	return(0);
}

int dealloca_memoria_device() {
	hipDeviceSynchronize();
        CUDASAFE( hipFree(hdev_allData) , "hipFree: &hdev_allData[]");
	return(0);
}



#define FLAGDEV_SOLVER_CU 1
#endif
